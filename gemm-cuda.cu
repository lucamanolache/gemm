
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define EPS 0.001

#define N 2048
#define BLOCK 16

__global__ void _gemm(float *a, float *b, float *c) {
  int row = blockIdx.y * BLOCK + threadIdx.y;
  int col = blockIdx.x * BLOCK + threadIdx.x;

  __shared__ float a_cache[BLOCK][BLOCK];
  __shared__ float b_cache[BLOCK][BLOCK];

  float dot = 0.0;
  for (int i = 0; i < N; i += BLOCK) {
    if (i + threadIdx.x < N && row < N)
      a_cache[threadIdx.y][threadIdx.x] = a[row * N + i + threadIdx.x];
    else
      a_cache[threadIdx.y][threadIdx.x] = 0;
    if (i + threadIdx.y < N && i + threadIdx.y < N)
      b_cache[threadIdx.y][threadIdx.x] = b[(i + threadIdx.y) * N + col];
    else
      b_cache[threadIdx.y][threadIdx.x] = 0;
    __syncthreads();

    for (int j = 0; j < BLOCK; j++) {
      dot += a_cache[threadIdx.y][j] * b_cache[j][threadIdx.x];
    }
    __syncthreads();
  }
  if (col < N && row < N)
    c[row * N + col] = dot;
}

void gemm(float *a_h, float *b_h, float *c_h) {
  float *a_d, *b_d, *c_d;

  hipMalloc(&a_d, N * N * sizeof(float));
  hipMalloc(&b_d, N * N * sizeof(float));
  hipMalloc(&c_d, N * N * sizeof(float));

  hipMemcpy(a_d, a_h, N * N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(b_d, b_h, N * N * sizeof(float), hipMemcpyHostToDevice);

  int numBlocks = N / BLOCK;

  float time_ms;
  hipEvent_t start, stop;

  // timing
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);

  _gemm<<<dim3(numBlocks, numBlocks, 1), dim3(BLOCK, BLOCK, 1)>>>(a_d, b_d,
                                                                  c_d);
  hipDeviceSynchronize();

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&time_ms, start, stop);

  printf("%f GFLOPS (compute)\n", (2.0 * N * N * N) / 1.0e6 / (double)time_ms);

  hipMemcpy(c_h, c_d, N * N * sizeof(float), hipMemcpyDeviceToHost);

  hipFree(a_d);
  hipFree(b_d);
  hipFree(c_d);
}

int main() {
  float *a = (float *)malloc(N * N * sizeof(float));
  float *b = (float *)malloc(N * N * sizeof(float));
  float *c = (float *)malloc(N * N * sizeof(float));
  float *ref = (float *)calloc(N * N, sizeof(float));

  FILE *f = fopen("gemm-out", "rb");
  fread(a, sizeof(float), N * N, f);
  fread(b, sizeof(float), N * N, f);
  fread(ref, sizeof(float), N * N, f);
  fclose(f);

  clock_t t;
  t = clock();
  gemm(a, b, c);
  t = clock() - t;
  double time_taken = ((double)t) / CLOCKS_PER_SEC; // in seconds
  printf("%f GFLOPS\n", (2.0 * N * N * N) / 1.0e9 / (double)time_taken);

  for (int i = 0; i < N * N; ++i) {
    if (fabs(c[i] - ref[i]) > EPS) {
      printf("expected %f, got %f at idx %d\n", ref[i], c[i], i);
      return -1;
    }
  }

  free(a);
  free(b);
  free(c);
  free(ref);

  return 0;
}
